
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop(int N)
{
  for (int i = 0; i < N; ++i)
  {
    printf("This is iteration number %d\n", i);
  }
}

__global__ void loop2(int N)
{
    
    
  int i = threadIdx.x + blockIdx.x * blockDim.x;
//this will print 1 time for each thread so it will print 9 times 0 + 0 * 0 = 0
// 1 + 0 * 0 = 1
// 2
  printf("This is iteration in parallel: %d\n", i);
  
}

__global__ void loop_for_single_block(int N)
{
    
    
  int i = threadIdx.x;
//this will print 1 time for each thread so it will print 9 times 0 + 0 * 0 = 0
// 1 + 0 * 0 = 1
// 2
  printf("This is iteration in parallel: %d\n", i);
  
}
int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, only use 1 block of threads.
   */

  int N = 10;
  //loop<<<1,9>>>(N);
    
  loop2<<<1,9>>>(N);
  hipDeviceSynchronize();
}
