
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
 * Refactor `loop` to be a CUDA Kernel. The new kernel should
 * only do the work of 1 iteration of the original loop.
 */

__global__ void loop(int N)
{
  int i = threadIdx.x + (blockIdx.x * blockDim.x);
  printf("This is iteration number %d\n", i);

 
}

int main()
{
  /*
   * When refactoring `loop` to launch as a kernel, be sure
   * to use the execution configuration to control how many
   * "iterations" to perform.
   *
   * For this exercise, be sure to use more than 1 block in
   * the execution configuration.
   */

  int N = 100;
  loop<<<10,10>>>(N);
  hipDeviceSynchronize();

}
