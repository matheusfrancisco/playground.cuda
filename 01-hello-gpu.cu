
#include <hip/hip_runtime.h>
#include <stdio.h>

void helloCPU()
{
  printf("Hello from the CPU.\n");
}

/*
 * Refactor the `helloGPU` definition to be a kernel
 * that can be launched on the GPU. Update its message
 * to read "Hello from the GPU!"
 * R: I should use __global__ and when I call this function I should pass the execution
 */

__global__ void helloGPU()
{
  printf("Hello also from the CPU.\n");
}

int main()
{

  helloCPU();

  /*
   * Refactor this call to `helloGPU` so that it launches
   * as a kernel on the GPU.
   * functionName<<number_of_block, number_of_threads>>  -> helloGPU<<1,1>> is 1 block 1 thread this mean
   */

  helloGPU<<<1,1>>>();
  hipDeviceSynchronize();

  /*
   * Add code below to synchronize on the completion of the
   * `helloGPU` kernel completion before continuing the CPU
   * thread.
   */
}
